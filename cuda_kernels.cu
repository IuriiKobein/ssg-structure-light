#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <opencv2/core/cuda.hpp>
#include "cuda_kernels.h"

__global__ void invert(cv::cuda::PtrStepSzf x, cv::cuda::PtrStepSzf y,
                       cv::cuda::PtrStepSzf z, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n) return;

    if (i % 2 == 0)
        z(0, i) = x(0, i / 2);
    else
        z(0, i) = y(0, (i - 1) / 2);
}

__global__ void invert2D(cv::cuda::PtrStepSzf x, cv::cuda::PtrStepSzf y,
                         cv::cuda::PtrStepSzf z) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= x.cols || j >= x.rows) return;

    if (j % 2 == 0)
        z(i, j) = x(i, j / 2);
    else
        z(i, j) = y(i, (j - 1) / 2);
}

__global__ void sin(cv::cuda::PtrStepSzf x, cv::cuda::PtrStepSzf y) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= x.cols || j >= x.rows) return;

    y(i, j) = sin(x(i, j));
}

__global__ void cos(cv::cuda::PtrStepSzf x, cv::cuda::PtrStepSzf y) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= x.cols || j >= x.rows) return;

    y(i, j) = cos(x(i, j));
}

__global__ void round(cv::cuda::PtrStepSzf x, cv::cuda::PtrStepSzf y) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= x.cols || j >= x.rows) return;

    y(i, j) = rint(x(i, j));
}

static __global__ void img_atan_inplace(cv::cuda::PtrStepSzf in0,
                                        cv::cuda::PtrStepSzf in1,
                                        cv::cuda::PtrStepSzf in2,
                                        cv::cuda::PtrStepSzf in3,
                                        cv::cuda::PtrStepSzf out) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in0.cols || y >= in0.rows) return;

    out(x, y) = atan2f(in3(x, y) - in1(x, y), in0(x, y) - in2(x, y));
}

static __global__ void dft2dct_inplace(cv::cuda::PtrStepSz<float2> in0,
                                       cv::cuda::PtrStepSzf cos_coeff,
                                       cv::cuda::PtrStepSzf sin_coeff,
                                       cv::cuda::PtrStepSzf out) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in0.cols || y >= in0.rows) return;

    out(x, y) = in0(x, y).x * cos_coeff(x, y);
    out(x, y) += in0(x, y).y * sin_coeff(x, y);
}

static __global__ void idft2idct_inplace(cv::cuda::PtrStepSzf in0,
                                         cv::cuda::PtrStepSzf cos_coeff,
                                         cv::cuda::PtrStepSzf sin_coeff,
                                         cv::cuda::PtrStepSz<float2> out) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in0.cols || y >= in0.rows) return;

    out(x, y).x = in0(x, y) * cos_coeff(x, y);
    out(x, y).y = in0(x, y) * sin_coeff(x, y);
}

static __global__ void delta_phi_inplace(cv::cuda::PtrStepSzf in0,
                                         cv::cuda::PtrStepSzf in1,
                                         cv::cuda::PtrStepSzf cos_coeff,
                                         cv::cuda::PtrStepSzf sin_coeff,
                                         cv::cuda::PtrStepSzf out) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= in0.cols || y >= in0.rows) return;

    out(x, y) = in0(x, y) * cos_coeff(x, y) - in1(x, y) * sin_coeff(x, y);
}

void invertArray(cv::cuda::GpuMat &x, cv::cuda::GpuMat &y,
                 cv::cuda::GpuMat &z) {
    const dim3 block(16, 16);
    const dim3 grid(cv::cudev::divUp(x.cols, block.x),
                    cv::cudev::divUp(x.rows, block.y));
    invert2D<<<grid, block>>>(x, y, z);
}

void cudaSin(cv::cuda::GpuMat &x, cv::cuda::GpuMat &y) {
    const dim3 block(16, 16);
    const dim3 grid(cv::cudev::divUp(x.cols, block.x),
                    cv::cudev::divUp(x.rows, block.y));
    sin<<<grid, block>>>(x, y);
}

void cudaCos(cv::cuda::GpuMat &x, cv::cuda::GpuMat &y) {
    const dim3 block(16, 16);
    const dim3 grid(cv::cudev::divUp(x.cols, block.x),
                    cv::cudev::divUp(x.rows, block.y));
    cos<<<grid, block>>>(x, y);
}

void cudaRound(cv::cuda::GpuMat &x, cv::cuda::GpuMat &y) {
    const dim3 block(16, 16);
    const dim3 grid(cv::cudev::divUp(x.cols, block.x),
                    cv::cudev::divUp(x.rows, block.y));
    round<<<grid, block>>>(x, y);
}

cv::cuda::GpuMat &cuda_diff_atan_inplace(
    std::vector<cv::cuda::GpuMat> &d_input) {
    const dim3 block(16, 16);

    const dim3 grid(cv::cudev::divUp(d_input[0].cols, block.x),
                    cv::cudev::divUp(d_input[0].rows, block.y));

    img_atan_inplace<<<grid, block>>>(d_input[0], d_input[1], d_input[2],
                                      d_input[3], d_input[0]);

    return d_input[0];
}

void cuda_dft2dct_out_convert(const cv::cuda::GpuMat &d_input,
                              const cv::cuda::GpuMat &d_cos_f,
                              const cv::cuda::GpuMat &d_sin_f,
                              cv::cuda::GpuMat &d_out) {
    const dim3 block(16, 16);

    const dim3 grid(cv::cudev::divUp(d_input.cols, block.x),
                    cv::cudev::divUp(d_input.rows, block.y));

    dft2dct_inplace<<<grid, block>>>(d_input, d_cos_f, d_sin_f, d_out);
}

void cuda_idft2idct_in_convert(const cv::cuda::GpuMat &d_input,
                               const cv::cuda::GpuMat &d_cos_f,
                               const cv::cuda::GpuMat &d_sin_f,
                               cv::cuda::GpuMat &d_out) {
    const dim3 block(16, 16);

    const dim3 grid(cv::cudev::divUp(d_input.cols, block.x),
                    cv::cudev::divUp(d_input.rows, block.y));

    idft2idct_inplace<<<grid, block>>>(d_input, d_cos_f, d_sin_f, d_out);
}

void cuda_delta_phi_mult_sub_inplace(cv::cuda::GpuMat &d_in1,
                                     cv::cuda::GpuMat &d_in2,
                                     const cv::cuda::GpuMat &d_cos_f,
                                     const cv::cuda::GpuMat &d_sin_f,
                                     cv::cuda::GpuMat &d_out) {
    const dim3 block(16, 16);

    const dim3 grid(cv::cudev::divUp(d_in1.cols, block.x),
                    cv::cudev::divUp(d_in1.rows, block.y));

    delta_phi_inplace<<<grid, block>>>(d_in1, d_in2, d_cos_f, d_sin_f, d_out);
}
